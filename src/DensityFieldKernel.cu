#include "hip/hip_runtime.h"
#include "GPUArray.cuh"
#include <hip/hip_runtime.h>
#include "DensityFieldKernel.h"

namespace DensityKernel
{
    __device__ void PBCDistance(real3& vec1, real3& vec2, real3& box, real3& dist, Real& dist_sq)
    {
        dist_sq=0.0;
        for (int i=0;i<3;i++){
            Real diff = vec1[i] - vec2[i];
            if (diff < (-0.5 * box[i])) {diff = diff + box[i];}
            else if (diff > (0.5  * box[i]))  {diff -= box[i];}
            dist_sq += (diff * diff);
            dist[i] = diff;
        }
    }

    __device__ Real GaussianKernel(Real rsq, Real inv_sigma2, Real prefactor)
    {
        return prefactor * expf(rsq * inv_sigma2);
    }

    __device__ void AddIndex(ind3& ind1, ind3& ind2, ind3& maxN, ind3& ret){
        for (int i=0;i<3;i++){
            ret[i] = (ind1[i] + ind2[i]) % maxN[i];

            if (ret[i] < 0){
                ret[i] += maxN[i];
            }
        }
    }

    // function on the device that converts positional index to lattice position
    __device__ void ConvertIndexToLatticePos(ind3& ind, real3& dx, real3& pos){
        for (int i=0;i<3;i++){
            pos[i] = ind[i] * dx[i];
        }
    }

    // function on the device that converts a 3d index into a 1d index
    __device__ void Convert3dIndexTo1d(ind3& index_3d, ind3& max_N, int& index_1d)
    {
        index_1d = index_3d[2] * max_N[0] * max_N[1] + index_3d[1] * max_N[0] + index_3d[0];
    }

    // function that sums the 2 vectors and stores it in C 
    __global__ void Sum(Real* A, Real* B, Real* C, int total_size){
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        if (idx < total_size){
            C[idx] = A[idx] + B[idx];
        }
    }

    __global__ void SelectiveSum(Real* A, Real* B, Real* C, int* indices, int total_size){
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        if (idx < total_size){
            int index = indices[idx];

            C[index] = A[index] + B[idx];
        }
    }

    __global__ void CalculateInstantaneousFieldKernel(Real* vector_field, Real* atom_positions, int* vector_field_index, \
                                                Real inv_sigma2, Real prefactor, Real* dx, Real* box, int* total_size, int* neighbors_index, int total_neighbors, int total_data)
    {
        // find the index of the current thread 
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        // if the index is smaller than the current data 
        if (idx < total_data){
            int atom_idx = idx / total_neighbors;
            int neighbor_idx = idx % total_neighbors;

            // copy over all the data
            ind3 pos_index;
            real3 pos_copy;
            ind3 total_size_copy;
            ind3 offset;
            real3 box_copy;
            real3 dx_copy;
            ind3 actual_index;
            real3 lattice_pos;
            real3 r_vec;
            Real dist_sq=0.0;
            int v_index;

            for (int i=0;i<3;i++){
                pos_copy[i] = atom_positions[atom_idx * 3 + i];
                pos_index[i] = floor(pos_copy[i] / dx[i]);
                box_copy[i] = box[i];
                offset[i] = neighbors_index[neighbor_idx * 3 + i];
                total_size_copy[i] = total_size[i];
                dx_copy[i] = dx[i];
            }

            // add index to 
            AddIndex(pos_index, offset, total_size_copy, actual_index);

            // convert 3d index to 1d 
            Convert3dIndexTo1d(actual_index, total_size_copy, v_index);

            // store the vector index 
            vector_field_index[idx] = v_index;

            // convert index to lattice position
            ConvertIndexToLatticePos(actual_index, dx_copy, lattice_pos);

            // calculate the periodic boundary condition distance 
            PBCDistance(pos_copy, lattice_pos, box_copy, r_vec, dist_sq);

            // calculate the gaussian weight and store into vector_field 
            float gaussWeight = GaussianKernel(dist_sq, inv_sigma2, prefactor);
            vector_field[idx] =  gaussWeight;
        }
        else{
            vector_field[idx] = 0.0;
            vector_field_index[idx] = 0;
        }
    }

}

void DensityKernel::CalculateInstantaneousField(GPUArray2d<Real>& vector_field_neighbors, GPUArray2d<Real>& atom_positions, GPUArray2d<int>& vector_field_neighbor_index, \
                                Real inv_sigma2, Real prefactor, GPUArray1d<Real>& box, GPUArray1d<Real>& dx, GPUArray1d<int>& N, \
                                    GPUArray3d<Real>& insta_field, GPUArray3d<Real>& field, GPUArray2d<int>& neighbor_index, int num_atoms, int num_thread)
{
    int NeighborSize = neighbor_index.getSize()[0];
    int TotalSize = NeighborSize * num_atoms;
    int numBlocks=(TotalSize + num_thread) / num_thread;

    // calculate the instantaneous field 
    thrust::fill(vector_field_neighbors.device_vector().begin(), vector_field_neighbors.device_vector().end(), 0.0);

    DensityKernel::CalculateInstantaneousFieldKernel<<<numBlocks, num_thread>>>(vector_field_neighbors.device_data(), atom_positions.device_data(), vector_field_neighbor_index.device_data(), \
                                                                inv_sigma2, prefactor, dx.device_data(), box.device_data(), N.device_data(), neighbor_index.device_data(), \
                                                                NeighborSize, TotalSize);
    thrust::sort_by_key(thrust::device, vector_field_neighbor_index.device_vector().begin(), vector_field_neighbor_index.device_vector().end(), \
                                        vector_field_neighbors.device_vector().begin());

    auto new_end = thrust::reduce_by_key(thrust::device, vector_field_neighbor_index.device_vector().begin(), vector_field_neighbor_index.device_vector().end(), \
                                        vector_field_neighbors.device_vector().begin(), vector_field_neighbor_index.device_vector().begin(), insta_field.device_vector().begin());
    int new_size = new_end.first - vector_field_neighbor_index.device_vector().begin();

    int num_field_Blocks = (new_size + num_thread) / num_thread;
    DensityKernel::SelectiveSum<<<num_field_Blocks, num_thread>>>(field.device_data(), insta_field.device_data(), field.device_data(), vector_field_neighbor_index.device_data(), new_size);
};

void DensityKernel::FillGPUArray(GPUArray3d<Real>& arr, Real num){
    thrust::fill(arr.device_vector().begin(), arr.device_vector().end(), num);
}
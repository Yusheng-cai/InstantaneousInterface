#include "hip/hip_runtime.h"
#include "GPUArray.h"
#include <hip/hip_runtime.h>

namespace DensityKernel
{
    using real3 = float[3];
    using ind3   = int[3];

    __device__ void PBCDistance(real3& vec1, real3& vec2, real3& box, real3& dist, float& dist_sq)
    {
        dist_sq=0.0;
        for (int i=0;i<3;i++){
            float diff = vec1[i] - vec2[i];
            if (diff < (-0.5 * box[i])) {diff = diff + box[i];}
            else if (diff > (0.5  * box[i]))  {diff -= box[i];}
            dist_sq += (diff * diff);
            dist[i] = diff;
        }
    }

    __device__ float GaussianKernel(float rsq, float inv_sigma2, float prefactor)
    {
        return prefactor * expf(rsq * inv_sigma2);
    }

    __device__ void AddIndex(ind3& ind1, ind3& ind2, ind3& maxN, ind3& ret){
        for (int i=0;i<3;i++){
            ret[i] = (ind1[i] + ind2[i]) % maxN[i];

            if (ret[i] < 0){
                ret[i] += maxN[i];
            }
        }
    }

    __device__ void ConvertIndexToLatticePos(ind3& ind, real3& dx, real3& pos){
        for (int i=0;i<3;i++){
            pos[i] = ind[i] * dx[i];
        }
    }

    __device__ void LocalIndexToNeighborOffset(int index, ind3& neighbors, ind3& offset){
        int xdim = 2 * neighbors[0] + 1;
        int ydim = 2 * neighbors[1] + 1;
        int xy = xdim * ydim;

        int zindex = index / xy; 
        int yindex = (index - zindex * xy) / xdim;
        int xindex = index - zindex * xy - yindex * xdim;

        offset[0] = xindex - neighbors[0];
        offset[1] = yindex - neighbors[1];
        offset[2] = zindex - neighbors[2];
    }

    __device__ void Convert3dIndexTo1d(ind3& index_3d, ind3& max_N, int& index_1d)
    {
        index_1d = index_3d[2] * max_N[0] * max_N[1] + index_3d[1] * max_N[0] + index_3d[0];
    }

    __global__ void Sum(float* A, float* B, float* C, int total_size){
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        if (idx < total_size){
            C[idx] = A[idx] + B[idx];
        }
    }

    __global__ void CalculateInstantaneousField(float* vector_field, float* atom_positions, int* vector_field_index, float inv_sigma2, float prefactor, float* dx, float* box, int* total_size, int* neighbors, int total_data)
    {
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        if (idx < total_data){
            int total_neighbors = (2 * neighbors[0] + 1) * (2 * neighbors[1] + 1) * (2 * neighbors[2] + 1);
            int atom_idx = idx / total_neighbors;
            int neighbor_idx = idx % total_neighbors;

            // // copy over the positions
            ind3 pos_index;
            real3 pos_copy;
            ind3 total_size_copy;
            ind3 neighbor_size_copy;
            real3 box_copy;
            real3 dx_copy;

            ind3 actual_index;
            real3 lattice_pos;
            real3 dist;
            float dist_sq;

            for (int i=0;i<3;i++){
                pos_copy[i] = atom_positions[atom_idx*3+i];
                pos_index[i] = floor(pos_copy[i] / dx[i]);
                box_copy[i] = box[i];
                neighbor_size_copy[i] = neighbors[i];
                total_size_copy[i] = total_size[i];
                dx_copy[i] = dx[i];
            }

            ind3 offset;
            int v_index;
            LocalIndexToNeighborOffset(neighbor_idx, neighbor_size_copy, offset);
            AddIndex(pos_index, offset, total_size_copy, actual_index);
            Convert3dIndexTo1d(actual_index, total_size_copy, v_index);
            vector_field_index[idx] = v_index;
            ConvertIndexToLatticePos(actual_index, dx_copy, lattice_pos);
            PBCDistance(pos_copy, lattice_pos, box_copy, dist, dist_sq);
            float gauss_weight = GaussianKernel(dist_sq, inv_sigma2, prefactor);
            vector_field[idx] = gauss_weight;
        }
        else{
            vector_field[idx] = 0.0;
            vector_field_index[idx] = 0;
        }
    }
}